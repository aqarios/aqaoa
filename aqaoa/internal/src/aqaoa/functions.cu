#include "hip/hip_runtime.h"
/* 
 * Modified by Aqarios GmbH
 * 
 * Changes: Renamed all occurrences of 'cuaoa' to 'aqaoa'.
 *
 * Original License Notice:
 *
 * Copyright 2024 Jonas Blenninger
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "aqaoa/functions.hpp"
#include "kernel/kernels.hpp"
#include <cfloat>
#include <cstdint>
#include <cstring>
#include <hip/hip_runtime.h>

hipDoubleComplex calcInitialValue(size_t numStates) {
  return make_hipDoubleComplex(1.0 / std::sqrt(numStates), 0.0);
}

void initializeHandle(AqAOAHandle *handle, size_t numNodes, size_t *polykeys,
                      double *polyvals, size_t polysize, size_t blockSize) {
  size_t numStates = 1 << numNodes;
  size_t numBlocks = (numStates + blockSize - 1) / blockSize;

  handle->mallocPoly(polykeys, polyvals, polysize);

  hipMemcpyAsync(handle->devicePolyKeys, polykeys, polysize * sizeof(size_t),
                  hipMemcpyHostToDevice, handle->stream);
  hipMemcpyAsync(handle->devicePolyVals, polyvals, polysize * sizeof(double),
                  hipMemcpyHostToDevice, handle->stream);
  handle->polykeys = polykeys;
  handle->polyvals = polyvals;
  handle->numNodes = numNodes;
  handle->numStates = 1 << numNodes;
  handle->polySize = polysize;
  handle->blockSize = blockSize;
  handle->numBlocks = numBlocks;
}

void initializeStatevector(AqAOAHandle *handle) {
  hipDoubleComplex initVal = calcInitialValue(handle->numStates);
  initializeSv(handle->stream, handle->numBlocks, handle->blockSize,
               handle->deviceStatevector, initVal, handle->numStates);
}

double calculateExpectationValue(AqAOAHandle *handle) {
  handle->mallocForExpval();
  double expval =
      calcExpval(handle->stream, handle->blockSize, handle->deviceStatevector,
                 handle->deviceCostHamiltonian, handle->deviceIntermediate,
                 handle->numStates);
  handle->freeForExpval();
  return expval;
}

hipDoubleComplex *retrieveStatevector(AqAOAHandle *handle) {
  hipDoubleComplex *hostStatevector;
  hipHostMalloc(&hostStatevector, handle->numStates * sizeof(hipDoubleComplex));
  hipMemcpyAsync(hostStatevector, handle->deviceStatevector,
                  handle->numStates * sizeof(hipDoubleComplex),
                  hipMemcpyDeviceToHost, handle->stream);
  hipStreamSynchronize(handle->stream);
  return hostStatevector;
}

double *retrieveCostHamiltonian(AqAOAHandle *handle) {
  double *hostCostHamiltonian;
  hipHostMalloc(&hostCostHamiltonian, handle->numStates * sizeof(double));
  hipMemcpyAsync(hostCostHamiltonian, handle->deviceCostHamiltonian,
                  handle->numStates * sizeof(double), hipMemcpyDeviceToHost,
                  handle->stream);
  return hostCostHamiltonian;
}

size_t *retrievePolykeys(AqAOAHandle *handle) {
  size_t *host;
  hipHostMalloc(&host, handle->polySize * sizeof(size_t));
  hipMemcpyAsync(host, handle->devicePolyKeys,
                  handle->polySize * sizeof(size_t), hipMemcpyDeviceToHost,
                  handle->stream);
  return host;
}

double *retrievePolyVals(AqAOAHandle *handle) {
  double *host;
  hipHostMalloc(&host, handle->polySize * sizeof(double));
  hipMemcpyAsync(host, handle->devicePolyVals,
                  handle->polySize * sizeof(double), hipMemcpyDeviceToHost,
                  handle->stream);
  return host;
}

void applyCostHamiltonian(AqAOAHandle *handle, double gamma) {
  applyHc(handle->stream, handle->numBlocks, handle->blockSize,
          handle->deviceStatevector, handle->deviceCostHamiltonian, gamma,
          handle->numStates);
}

void applyRxGate(AqAOAHandle *handle, double beta, int32_t adjoint,
                 const char *method) {
  if (strcmp(method, "custatevec") == 0) {
    applyRxGate(handle, beta, adjoint);
  } else if (strcmp(method, "qokit") == 0) {
    applyRxGateQO(handle, beta, adjoint);
  } else {
    applyRxGate(handle, beta, adjoint);
  }
}

void applyRxGate(AqAOAHandle *handle, double beta, int32_t adjoint) {
  applyRxGate(handle->handle, beta, handle->deviceStatevector, adjoint,
              handle->numNodes);
}

void applyRxGateQO(AqAOAHandle *handle, double beta, int32_t adjoint) {
  applyRxGateQO(handle->stream, handle->numBlocks, handle->blockSize, beta,
                handle->deviceStatevector, adjoint, handle->numNodes);
}

void calcCostHamiltonian(AqAOAHandle *handle) {
  calcHc(handle->stream, handle->numBlocks, handle->blockSize,
         handle->deviceCostHamiltonian, handle->devicePolyKeys,
         handle->devicePolyVals, handle->polySize, handle->numStates);
}

SampleSet sampleStatevector(AqAOAHandle *handle, uint32_t maxShots,
                            const uint32_t numShots, const double *randnums) {
  custatevecSamplerDescriptor_t sampler;
  size_t extraWorkspaceSizeInBytes = 0;
  custatevecStatus_t status_a = custatevecSamplerCreate(
      handle->handle, handle->deviceStatevector, HIP_C_64F, handle->numNodes,
      &sampler, maxShots, &extraWorkspaceSizeInBytes);

  void *extraWorkspace = nullptr;
  if (extraWorkspaceSizeInBytes > 0) {
    hipMallocAsync((void **)&extraWorkspace, extraWorkspaceSizeInBytes,
                    handle->stream);
  }

  custatevecStatus_t status_b = custatevecSamplerPreprocess(
      handle->handle, sampler, extraWorkspace, extraWorkspaceSizeInBytes);

  SampleSet sample_set;
  sample_set.bitStrings = new int64_t[numShots];

  int32_t *bitOrderingT = new int32_t[handle->numNodes];
  for (int32_t i = 0; i < handle->numNodes; i++) {
    bitOrderingT[i] = i;
  }
  const int32_t *bitOrdering = bitOrderingT;
  custatevecSamplerOutput_t output = CUSTATEVEC_SAMPLER_OUTPUT_RANDNUM_ORDER;

  custatevecStatus_t status_c = custatevecSamplerSample(
      handle->handle, sampler, (custatevecIndex_t *)sample_set.bitStrings,
      bitOrdering, handle->numNodes, randnums, numShots, output);

  custatevecStatus_t status_d = custatevecSamplerDestroy(sampler);
  hipFreeAsync(extraWorkspace, handle->stream);

  double *d_energies;
  int64_t *d_samples;

  hipMallocAsync((void **)&d_energies, numShots * sizeof(double),
                  handle->stream);
  hipMallocAsync((void **)&d_samples, numShots * sizeof(int64_t),
                  handle->stream);

  hipMemcpyAsync(d_samples, sample_set.bitStrings, numShots * sizeof(int64_t),
                  hipMemcpyHostToDevice, handle->stream);

  findEnergies(handle->stream, handle->numBlocks, handle->blockSize,
               handle->deviceCostHamiltonian, d_samples, d_energies, numShots);

  sample_set.energies = new double[numShots];
  hipMemcpyAsync(sample_set.energies, d_energies, numShots * sizeof(double),
                  hipMemcpyDeviceToHost, handle->stream);
  hipFreeAsync(d_energies, handle->stream);
  hipFreeAsync(d_samples, handle->stream);

  return sample_set;
}

void calculateGradients(AqAOAHandle *handle, size_t depth, const double *betas,
                        const double *gammas, double *betaGradients,
                        double *gammaGradients, const char *rxmethod) {
  hipStreamSynchronize(handle->stream);
  copyC(handle->stream, handle->numBlocks, handle->blockSize,
        handle->deviceStatevector, handle->devicePhi, handle->numStates);
  mulH(handle->stream, handle->numBlocks, handle->blockSize,
       handle->deviceStatevector, handle->deviceCostHamiltonian,
       handle->numStates);

  double *pinnedHostGrad;
  hipHostMalloc(&pinnedHostGrad, sizeof(double));

  for (int p = depth - 1; p >= 0; p--) {
    betaGradients[p] = 0.0;
    for (int q = 0; q < handle->numNodes; q++) {
      handle->resetGradientHolder();
      applyXGateSingle(handle->handle, handle->devicePhi, q, handle->numNodes);
      calcInnerProd(handle->stream, handle->numBlocks, handle->blockSize,
                    handle->deviceStatevector, handle->devicePhi,
                    handle->deviceGradientHolder, handle->numStates);
      applyXGateSingle(handle->handle, handle->devicePhi, q, handle->numNodes);
      hipMemcpyAsync(pinnedHostGrad, handle->deviceGradientHolder,
                      sizeof(double), hipMemcpyDeviceToHost, handle->stream);
      hipStreamSynchronize(handle->stream);
      betaGradients[p] -= *pinnedHostGrad;
    }
    betaGradients[p] *= -2.0;

    if (rxmethod == "custatevec") {
      applyRxGate(handle->handle, betas[p], handle->deviceStatevector, 1,
                  handle->numNodes);
      applyRxGate(handle->handle, betas[p], handle->devicePhi, 1,
                  handle->numNodes);
    } else if (rxmethod == "qokit") {
      applyRxGateQO(handle->stream, handle->numBlocks, handle->blockSize,
                    betas[p], handle->deviceStatevector, 1, handle->numNodes);
      applyRxGateQO(handle->stream, handle->numBlocks, handle->blockSize,
                    betas[p], handle->devicePhi, 1, handle->numNodes);
    } else {
      applyRxGate(handle->handle, betas[p], handle->deviceStatevector, 1,
                  handle->numNodes);
      applyRxGate(handle->handle, betas[p], handle->devicePhi, 1,
                  handle->numNodes);
    }
    handle->resetGradientHolder();

    calcZZGrad(handle->stream, handle->numBlocks, handle->blockSize,
               handle->deviceStatevector, handle->devicePhi,
               handle->deviceCostHamiltonian, handle->deviceGradientHolder,
               handle->numStates);

    hipMemcpyAsync(pinnedHostGrad, handle->deviceGradientHolder,
                    sizeof(double), hipMemcpyDeviceToHost, handle->stream);
    hipStreamSynchronize(handle->stream);
    gammaGradients[p] = -2.0 * *pinnedHostGrad;

    if (p > 0) {
      applyHc(handle->stream, handle->numBlocks, handle->blockSize,
              handle->deviceStatevector, handle->deviceCostHamiltonian,
              -gammas[p], handle->numStates);
      applyHc(handle->stream, handle->numBlocks, handle->blockSize,
              handle->devicePhi, handle->deviceCostHamiltonian, -gammas[p],
              handle->numStates);
    }
    hipStreamSynchronize(handle->stream);
  }
}

int64_t *findMinima(AqAOAHandle *handle, double *minimum, size_t *numMatches) {
  *minimum = DBL_MAX;
  *numMatches = 0;

  int64_t *d_minIndices;
  int *d_numMatches;
  double *d_minCost;

  hipMallocAsync(&d_minIndices, handle->numStates * sizeof(int64_t),
                  handle->stream);
  hipMallocAsync(&d_numMatches, sizeof(int), handle->stream);
  hipMallocAsync(&d_minCost, sizeof(double), handle->stream);

  hipMemcpyAsync(d_minCost, minimum, sizeof(double), hipMemcpyHostToDevice,
                  handle->stream);
  hipMemcpyAsync(d_numMatches, numMatches, sizeof(int), hipMemcpyHostToDevice,
                  handle->stream);

  findMinimum(handle->stream, handle->numBlocks, handle->blockSize,
              handle->deviceCostHamiltonian, d_minCost, handle->numStates);

  hipMemcpyAsync(minimum, d_minCost, sizeof(double), hipMemcpyDeviceToHost,
                  handle->stream);

  findIndicesOfMinimum(handle->stream, handle->numBlocks, handle->blockSize,
                       handle->deviceCostHamiltonian, *minimum, d_minIndices,
                       d_numMatches, handle->numStates);

  int numM;
  hipMemcpyAsync(&numM, d_numMatches, sizeof(int), hipMemcpyDeviceToHost,
                  handle->stream);

  *numMatches = (size_t)numM;
  int64_t *bitStrings;
  hipHostMalloc(&bitStrings, *numMatches * sizeof(int64_t));
  hipMemcpyAsync(bitStrings, d_minIndices, *numMatches * sizeof(int64_t),
                  hipMemcpyDeviceToHost, handle->stream);

  hipFreeAsync(d_minIndices, handle->stream);
  hipFreeAsync(d_numMatches, handle->stream);
  hipFreeAsync(d_minCost, handle->stream);

  hipStreamSynchronize(handle->stream);

  return bitStrings;
}
