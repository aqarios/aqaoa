#include "hip/hip_runtime.h"
/* 
 * Modified by Aqarios GmbH
 * 
 * Changes: Renamed all occurrences of 'cuaoa' to 'aqaoa'.
 *
 * Original License Notice:
 *
 * Copyright 2024 Jonas Blenninger
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "kernel/kernels.hpp"
#include <cfloat>
#include <cstddef>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>

__global__ void applyHcKernel(hipDoubleComplex *statevector, double *hamiltonian,
                              double gamma, size_t N) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    double angle = gamma * hamiltonian[i];
    hipDoubleComplex multiplier = make_hipDoubleComplex(cos(angle), sin(angle));
    statevector[i] = hipCmul(statevector[i], multiplier);
  }
}

void applyHc(hipStream_t stream, size_t numBlocks, size_t blockSize,
             hipDoubleComplex *statevector, double *hamiltonian, double gamma,
             size_t N) {
  applyHcKernel<<<numBlocks, blockSize, 0, stream>>>(statevector, hamiltonian,
                                                     gamma, N);
}

__global__ void copyCKernel(hipDoubleComplex *statevector,
                            hipDoubleComplex *other, size_t N) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    other[i] = statevector[i];
  }
}

void copyC(hipStream_t stream, size_t numBlocks, size_t blockSize,
           hipDoubleComplex *statevector, hipDoubleComplex *other, size_t N) {
  copyCKernel<<<numBlocks, blockSize, 0, stream>>>(statevector, other, N);
}

__global__ void calcHcKernel(double *diag, size_t *polykeys, double *polyvalues,
                             size_t polysize, size_t N) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    double buffer = 0.0;
    for (size_t j = 0; j < polysize; j++) {
      buffer += ((i & polykeys[j]) == polykeys[j]) * polyvalues[j];
    }
    diag[i] = buffer;
  }
}

void calcHc(hipStream_t stream, size_t numBlocks, size_t blockSize,
            double *diag, size_t *polykeys, double *polyvals, size_t polysize,
            size_t N) {
  calcHcKernel<<<numBlocks, blockSize, 0, stream>>>(diag, polykeys, polyvals,
                                                    polysize, N);
}

__global__ void findValueForIndexKernel(double *d_h, int64_t *d_samples,
                                        double *d_energies, size_t N) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    d_energies[i] = d_h[d_samples[i]];
  }
}

void findValueForIndex(hipStream_t stream, size_t numBlocks, size_t blockSize,
                       double *d_h, int64_t *d_samples, double *d_energies,
                       size_t N) {
  findValueForIndexKernel<<<numBlocks, blockSize, 0, stream>>>(d_h, d_samples,
                                                               d_energies, N);
}

__global__ void initializeSvKernel(hipDoubleComplex *statevector,
                                   hipDoubleComplex initialValue, size_t N) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    statevector[i] = initialValue;
  }
}

void initializeSv(hipStream_t stream, size_t numBlocks, size_t blockSize,
                  hipDoubleComplex *statevector, hipDoubleComplex initialValue,
                  size_t N) {
  initializeSvKernel<<<numBlocks, blockSize, 0, stream>>>(statevector,
                                                          initialValue, N);
}

__global__ void calcInnerProdKernel(hipDoubleComplex *sv, hipDoubleComplex *phi,
                                    double *gradientImag, size_t N) {
  extern __shared__ hipDoubleComplex temp[];

  int tid = threadIdx.x;
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < N) {
    hipDoubleComplex a = sv[index];
    hipDoubleComplex aconj = make_hipDoubleComplex(hipCreal(a), -hipCimag(a));
    hipDoubleComplex b = phi[index];
    temp[tid] = hipCmul(aconj, b);
  } else {
    temp[tid] = make_hipDoubleComplex(0.0, 0.0);
  }
  __syncthreads();

  for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      hipDoubleComplex elem = temp[tid];
      hipDoubleComplex o = temp[tid + s];
      hipDoubleComplex n = make_hipDoubleComplex(hipCreal(elem) + hipCreal(o),
                                               hipCimag(elem) + hipCimag(o));
      temp[tid] = n;
    }
    __syncthreads();
  }

  if (tid == 0) {
    atomicAdd(gradientImag, hipCimag(temp[0]));
  }
}

void calcInnerProd(hipStream_t stream, size_t numBlocks, size_t blockSize,
                   hipDoubleComplex *sv, hipDoubleComplex *phi, double *o,
                   size_t N) {
  size_t sharedMemSize = blockSize * sizeof(hipDoubleComplex);
  calcInnerProdKernel<<<numBlocks, blockSize, sharedMemSize, stream>>>(sv, phi,
                                                                       o, N);
}

__global__ void mulHKernel(hipDoubleComplex *statevector, double *hamiltonian,
                           size_t N) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    hipDoubleComplex v = statevector[i];
    double h = hamiltonian[i];
    statevector[i] = make_hipDoubleComplex(hipCreal(v) * h, hipCimag(v) * h);
  }
}

void mulH(hipStream_t stream, size_t numBlocks, size_t blockSize,
          hipDoubleComplex *statevector, double *hamiltonian, size_t N) {
  mulHKernel<<<numBlocks, blockSize, 0, stream>>>(statevector, hamiltonian, N);
}

__global__ void rxKernel(hipDoubleComplex *sv, size_t q, size_t N, double beta) {
  // Based on RX implementation from https://github.com/jpmorganchase/QOKit
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= (N / 2)) {
    return;
  }
  hipDoubleComplex cosang = make_hipDoubleComplex(cos(beta), 0.0);
  hipDoubleComplex sinang = make_hipDoubleComplex(0.0, -sin(beta));

  size_t m1 = (1 << q) - 1;
  size_t m2 = m1 ^ ((N - 1) >> 1);
  size_t ia = (i & m1) | ((i & m2) << 1);
  size_t ib = ia | (1 << q);

  hipDoubleComplex ta = sv[ia];
  hipDoubleComplex tb = sv[ib];

  hipDoubleComplex naA = hipCmul(ta, cosang);
  hipDoubleComplex naB = hipCmul(tb, sinang);
  hipDoubleComplex na = make_hipDoubleComplex(hipCreal(naA) + hipCreal(naB),
                                            hipCimag(naA) + hipCimag(naB));

  hipDoubleComplex nbA = hipCmul(ta, sinang);
  hipDoubleComplex nbB = hipCmul(tb, cosang);
  hipDoubleComplex nb = make_hipDoubleComplex(hipCreal(nbA) + hipCreal(nbB),
                                            hipCimag(nbA) + hipCimag(nbB));

  sv[ia] = na;
  sv[ib] = nb;
}

void applyRxGateQO(hipStream_t stream, size_t numBlocks, size_t blockSize,
                   double beta, hipDoubleComplex *d_sv, int32_t adjoint,
                   size_t numQubits) {
  double b = adjoint == 0 ? beta : -beta;
  size_t N = 1 << numQubits;
  numBlocks = ((N / 2) + blockSize - 1) / blockSize;
  for (size_t q = 0; q < numQubits; ++q) {
    rxKernel<<<numBlocks, blockSize, 0, stream>>>(d_sv, q, N, b);
  }
}

void applyRxGate(custatevecHandle_t handle, double beta, hipDoubleComplex *d_sv,
                 int32_t adjoint, size_t N) {
  hipDoubleComplex RX_MATRIX[] = {
      {std::cos(beta), 0.0},
      {0.0, -std::sin(beta)},
      {0.0, -std::sin(beta)},
      {std::cos(beta), 0.0},
  };

  for (int i = 0; i < N; i++) {
    int targets[] = {i};
    custatevecApplyMatrix(handle, d_sv, HIP_C_64F, N, RX_MATRIX, HIP_C_64F,
                          CUSTATEVEC_MATRIX_LAYOUT_ROW, adjoint, targets, 1,
                          nullptr, nullptr, 0, CUSTATEVEC_COMPUTE_64F, nullptr,
                          0);
  }
}

void applyXGate(custatevecHandle_t handle, hipDoubleComplex *d_sv, size_t N) {
  hipDoubleComplex X_MATRIX[] = {{0.0, 0.0}, {1.0, 0.0}, {1.0, 0.0}, {0.0, 0.0}};
  for (int i = 0; i < N; i++) {
    int targets[] = {i};
    custatevecApplyMatrix(handle, d_sv, HIP_C_64F, N, X_MATRIX, HIP_C_64F,
                          CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets, 1, nullptr,
                          nullptr, 0, CUSTATEVEC_COMPUTE_64F, nullptr, 0);
  }
}

void applyXGateSingle(custatevecHandle_t handle, hipDoubleComplex *d_sv,
                      int qubit, size_t N) {
  hipDoubleComplex X_MATRIX[] = {{0.0, 0.0}, {1.0, 0.0}, {1.0, 0.0}, {0.0, 0.0}};
  int targets[] = {qubit};
  custatevecApplyMatrix(handle, d_sv, HIP_C_64F, N, X_MATRIX, HIP_C_64F,
                        CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets, 1, nullptr,
                        nullptr, 0, CUSTATEVEC_COMPUTE_64F, nullptr, 0);
}

__global__ void calcZZGradKernel(hipDoubleComplex *sv, hipDoubleComplex *phi,
                                 double *d_h, double *gradientImag, size_t N) {
  extern __shared__ hipDoubleComplex temp[];

  int tid = threadIdx.x;
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < N) {
    hipDoubleComplex a = sv[index];
    hipDoubleComplex aconj =
        make_hipDoubleComplex(hipCreal(a) * d_h[index], -hipCimag(a) * d_h[index]);
    hipDoubleComplex b = phi[index];
    temp[tid] = hipCmul(aconj, b);
  } else {
    temp[tid] = make_hipDoubleComplex(0.0, 0.0);
  }
  __syncthreads();

  for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      hipDoubleComplex elem = temp[tid];
      hipDoubleComplex o = temp[tid + s];
      hipDoubleComplex n = make_hipDoubleComplex(hipCreal(elem) + hipCreal(o),
                                               hipCimag(elem) + hipCimag(o));
      temp[tid] = n;
    }
    __syncthreads();
  }

  if (tid == 0) {
    atomicAdd(gradientImag, hipCimag(temp[0]));
  }
}

void calcZZGrad(hipStream_t stream, size_t numBlocks, size_t blockSize,
                hipDoubleComplex *sv, hipDoubleComplex *phi, double *h, double *o,
                size_t N) {
  size_t sharedMemSize = blockSize * sizeof(hipDoubleComplex);
  calcZZGradKernel<<<numBlocks, blockSize, sharedMemSize, stream>>>(sv, phi, h,
                                                                    o, N);
}

__global__ void findEnergiesKernel(double *d_h, int64_t *d_samples,
                                   double *d_energies, uint32_t numShots) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < numShots) {
    d_energies[i] = d_h[d_samples[i]];
  }
}

void findEnergies(hipStream_t stream, size_t numBlocks, size_t blockSize,
                  double *d_h, int64_t *d_samples, double *d_energies,
                  uint32_t numShots) {
  findEnergiesKernel<<<numBlocks, blockSize, 0, stream>>>(d_h, d_samples,
                                                          d_energies, numShots);
}

__device__ double atomicMin_double(double *address, double val) {
  unsigned long long int *address_as_ull = (unsigned long long int *)address;
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(
        address_as_ull, assumed,
        __double_as_longlong(min(val, __longlong_as_double(assumed))));
  } while (assumed != old);
  return __longlong_as_double(old);
}

__global__ void findMinimumKernel(double *diag, double *minVal, size_t N) {
  extern __shared__ double sdata[];

  size_t tid = threadIdx.x;
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;

  sdata[tid] = (i < N) ? diag[i] : DBL_MAX;
  __syncthreads();

  for (size_t s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      sdata[tid] = min(sdata[tid], sdata[tid + s]);
    }
  }

  if (tid == 0)
    atomicMin_double(minVal, sdata[0]);
}

void findMinimum(hipStream_t stream, size_t numBlocks, size_t blockSize,
                 double *d_h, double *d_minCost, size_t N) {
  findMinimumKernel<<<numBlocks, blockSize, blockSize * sizeof(double),
                      stream>>>(d_h, d_minCost, N);
}

__global__ void findIndicesOfMinKernel(double *diag, double minVal,
                                       int64_t *minIndices, int *numMatches,
                                       size_t N) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    if (diag[i] == minVal) {
      int index = atomicAdd(numMatches, 1);
      minIndices[index] = i;
    }
  }
}

void findIndicesOfMinimum(hipStream_t stream, size_t numBlocks,
                          size_t blockSize, double *d_h, double minVal,
                          int64_t *d_minIndices, int *d_numMatches, size_t N) {
  findIndicesOfMinKernel<<<numBlocks, blockSize, 0, stream>>>(
      d_h, minVal, d_minIndices, d_numMatches, N);
}
